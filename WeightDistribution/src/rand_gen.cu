#include "hip/hip_runtime.h"
// Copyright 2016 Matthias Werner
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
// http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "globals.h"
#include "rand_gen.cuh"

#include <helper.h>

#define OFFSET_MULT 8137 // should be a higher number
/*
Notes:
i * 32 * curandDirectionVectors32 === hiprandDirectionVectors32_t[i]

offset in hiprand_init should be high between threads (e.g. offset=tid*XXXXX)
(first 'random' numbers are nonsense)

does every threads needs his own dimension aka own direction vector (aka own stream) ?
=> you are limited to 20k threads
=> better to play with offsets
(1 random number stream has sequence of sobol32 is 2^32)

hiprand() does not deliver random number (but index of next element), use curand_*() instead

better to use fewer threads with longer runnings within random number stream
 */

template<uint_t DIM>
__global__ void init_rand_gen(hiprandState_t *state,
    uint_t seed)
{
  uint_t tid = DIM*(threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x));

  hiprand_init(seed,
              tid,
              1,
              &state[tid]);
  if(DIM==2)
    hiprand_init(seed+1,
              tid+1,
              2,
              &state[tid]);
  if(DIM==3)
    hiprand_init(seed+2,
              tid+2,
              3,
              &state[tid]);
}


template<uint_t DIM, typename curandDirectionVectors_sz, typename curandStateSobol_sz>
__global__ void init_rand_gen(curandDirectionVectors_sz * sobolDirectionVectors,
    curandStateSobol_sz *state,
    uint_t offset)
{
  //uint_t tid = DIM*(threadIdx.x + blockDim.x * blockIdx.x);
  uint_t tid = DIM*(threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x));

  hiprand_init(sobolDirectionVectors[0],
              (offset+tid)*OFFSET_MULT,
              &state[tid]);
  if(DIM==2)
    hiprand_init(sobolDirectionVectors[1], // sobol direction vectors consists of 32 vectors (hiprandStateScrambledSobol32)
              (offset+tid)*OFFSET_MULT,
              &state[tid+1]);
  if(DIM==3)
      hiprand_init(sobolDirectionVectors[2], // sobol direction vectors consists of 32 vectors (hiprandStateScrambledSobol32)
              (offset+tid)*OFFSET_MULT,
              &state[tid+2]);
}

template<uint_t DIM>
__global__ void init_rand_gen(hiprandDirectionVectors32_t * sobolDirectionVectors,
    uint_t *sobolScrambleConstants,
    hiprandStateScrambledSobol32_t *state,
    uint_t offset)
{
  //uint_t tid = DIM*(threadIdx.x + blockDim.x * blockIdx.x);
  uint_t tid = DIM*(threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x));

  hiprand_init(sobolDirectionVectors[0],
              sobolScrambleConstants[0],
              (offset+tid)*OFFSET_MULT,
              &state[tid]);
  if(DIM==2)
    hiprand_init(sobolDirectionVectors[1], // sobol direction vectors consists of 32 vectors (hiprandStateScrambledSobol32)
              sobolScrambleConstants[1],
              (offset+tid)*OFFSET_MULT,
              &state[tid+1]);
  if(DIM==3)
      hiprand_init(sobolDirectionVectors[2], // sobol direction vectors consists of 32 vectors (hiprandStateScrambledSobol32)
              sobolScrambleConstants[2],
              (offset+tid)*OFFSET_MULT,
              &state[tid+2]);
}


void RandGen<hiprandStateScrambledSobol32_t>::init(dim3 blocks, dim3 threads, uint_t offset, uint_t dim, uint_t dev)
{
  uint_t max_threads = threads.x * blocks.x * blocks.y;

  CHECK_ERROR(hipSetDevice(dev));
  CHECK_ERROR_LIB(hiprandGetDirectionVectors32( &hostVectors, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6));
  CHECK_ERROR_LIB(hiprandGetScrambleConstants32( &hostScrambleConstants));

  CHECK_ERROR(hipMalloc(&devStates, dim*max_threads * sizeof(hiprandStateScrambledSobol32_t)));
  CHECK_ERROR(hipMalloc(&devDirectionVectors, dim * sizeof(hiprandDirectionVectors32_t)));
  CHECK_ERROR(hipMalloc(&devScrambleConstants, dim * sizeof(uint_t)));

  CHECK_ERROR(hipMemcpy(devDirectionVectors, hostVectors, dim * sizeof(hiprandDirectionVectors32_t),
                        hipMemcpyHostToDevice));
  CHECK_ERROR(hipMemcpy(devScrambleConstants, hostScrambleConstants, dim * sizeof(uint_t),
                        hipMemcpyHostToDevice));

  switch(dim)
  {
    case 2:
      init_rand_gen<2><<<blocks, threads>>>(devDirectionVectors, devScrambleConstants, devStates, offset);
      break;
    case 3:
      init_rand_gen<3><<<blocks, threads>>>(devDirectionVectors, devScrambleConstants, devStates, offset);
      break;
    default:
    init_rand_gen<1><<<blocks, threads>>>(devDirectionVectors, devScrambleConstants, devStates, offset);
  }
}

void RandGen<hiprandStateScrambledSobol32_t>::free()
{
  CHECK_ERROR(hipFree(devStates));
  CHECK_ERROR(hipFree(devDirectionVectors));
  CHECK_ERROR(hipFree(devScrambleConstants));
}

// --------

void RandGen<hiprandStateSobol32_t>::init(dim3 blocks, dim3 threads, uint_t offset, uint_t dim, uint_t dev)
{
  uint_t max_threads = threads.x * blocks.x * blocks.y;

  CHECK_ERROR(hipSetDevice(dev));
  CHECK_ERROR_LIB(hiprandGetDirectionVectors32( &hostVectors, HIPRAND_DIRECTION_VECTORS_32_JOEKUO6));
  CHECK_ERROR(hipMalloc(&devStates, dim*max_threads * sizeof(hiprandStateSobol32_t)));
  CHECK_ERROR(hipMalloc(&devDirectionVectors, dim * sizeof(hiprandDirectionVectors32_t)));
  CHECK_ERROR(hipMemcpy(devDirectionVectors, hostVectors, dim * sizeof(hiprandDirectionVectors32_t),
                        hipMemcpyHostToDevice));

  switch(dim)
  {
    case 2:
      init_rand_gen<2><<<blocks, threads>>>(devDirectionVectors, devStates, offset);
      break;
    case 3:
      init_rand_gen<3><<<blocks, threads>>>(devDirectionVectors, devStates, offset);
      break;
    default:
    init_rand_gen<1><<<blocks, threads>>>(devDirectionVectors, devStates, offset);
  }
}

void RandGen<hiprandStateSobol32_t>::free()
{
  CHECK_ERROR(hipFree(devStates));
  CHECK_ERROR(hipFree(devDirectionVectors));
}

// --------

void RandGen<hiprandStateSobol64_t>::init(dim3 blocks, dim3 threads, uint_t offset, uint_t dim, uint_t dev)
{
  uint_t max_threads = threads.x * blocks.x * blocks.y;

  CHECK_ERROR(hipSetDevice(dev));
  CHECK_ERROR_LIB(hiprandGetDirectionVectors64( &hostVectors, HIPRAND_DIRECTION_VECTORS_64_JOEKUO6));
  CHECK_ERROR(hipMalloc(&devStates, dim*max_threads * sizeof(hiprandStateSobol64_t)));
  CHECK_ERROR(hipMalloc(&devDirectionVectors, dim * sizeof(hiprandDirectionVectors64_t)));
  CHECK_ERROR(hipMemcpy(devDirectionVectors, hostVectors, dim * sizeof(hiprandDirectionVectors64_t),
                        hipMemcpyHostToDevice));

  switch(dim)
  {
    case 2:
      init_rand_gen<2><<<blocks, threads>>>(devDirectionVectors, devStates, offset);
      break;
    case 3:
      init_rand_gen<3><<<blocks, threads>>>(devDirectionVectors, devStates, offset);
      break;
    default:
    init_rand_gen<1><<<blocks, threads>>>(devDirectionVectors, devStates, offset);
  }
}

void RandGen<hiprandStateSobol64_t>::free()
{
  CHECK_ERROR(hipFree(devStates));
  CHECK_ERROR(hipFree(devDirectionVectors));
}

// --------
template<>
void RandGen<hiprandState_t>::init(dim3 blocks, dim3 threads, uint_t seed, uint_t dim, uint_t dev)
{
  uint_t max_threads = threads.x * blocks.x * blocks.y;

  CHECK_ERROR(hipSetDevice(dev));
  CHECK_ERROR(hipMalloc(&devStates, dim*max_threads * sizeof(hiprandStateSobol32_t)));

  switch(dim)
  {
    case 2:
      init_rand_gen<2><<<blocks, threads>>>(devStates, seed);
      break;
    case 3:
      init_rand_gen<3><<<blocks, threads>>>(devStates, seed);
      break;
    default:
    init_rand_gen<1><<<blocks, threads>>>(devStates, seed);
  }
}

template<>
void RandGen<hiprandState_t>::free()
{
  CHECK_ERROR(hipFree(devStates));
}
