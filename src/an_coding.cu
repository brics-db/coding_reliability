#include "hip/hip_runtime.h"

#include "globals.h"
#include "algorithms.h"
#include <helper.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#include <iostream>
#include <ostream>
using namespace std;

template<uintll_t ShardSize,uint_t CountCounts>
__global__
void dancoding(uintll_t n, uintll_t A, uintll_t* counts, uintll_t offset, uintll_t end)
{
  uintll_t shardXid = threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + offset;
  if(shardXid>=end)
    return;

  uintll_t counts_local[CountCounts] = { 0 };

  uintll_t w = A * shardXid * ShardSize;
  uintll_t wend = A * (shardXid+1) * ShardSize;
  uintll_t v;
  const uintll_t vend = (A<<n);
  for(;w<wend;w+=A)
  {
    for(v=w+A; v<vend; v+=A)
      ++counts_local[ __popcll( w^v ) ];
  }
  for(int c=1; c<CountCounts; ++c)
    atomicAdd(counts+c, counts_local[c]);
}

void run_ancoding(uintll_t n, uintll_t A, int verbose, uintll_t* minb, uintll_t* mincb, int file_output, int nr_dev_max)
{
  int tmp_nr_dev;
  Statistics stats;
  TimeStatistics results_cpu (&stats,CPU_WALL_TIME);
  TimeStatistics results_gpu (&stats,GPU_TIME);
  int i_runtime = results_gpu.add("Kernel Runtime", "s");
  int i_totaltime = results_cpu.add("Total Runtime", "s");
  results_cpu.setFactorAll(0.001);
  results_gpu.setFactorAll(0.001);


  CHECK_ERROR( hipGetDeviceCount(&tmp_nr_dev) );
  const int nr_dev = nr_dev_max==0 ? tmp_nr_dev : min(nr_dev_max,tmp_nr_dev);
  hipDeviceProp_t prop;
  if(verbose){
    printf("Start AN-Coding Algorithm\n");
    printf("Found %d CUDA devices.\n", nr_dev);
  }

  // skip init time
  for(int dev=0; dev<nr_dev; ++dev)
  {
    CHECK_ERROR( hipSetDevice(dev) );
    CHECK_ERROR( hipDeviceSynchronize() );
  }

  results_cpu.start(i_totaltime);

  const uintll_t count_messages = (1ull << n);
  const uintll_t size_shards = n<=8 ? 1 : n<=16 ? 16 : n<=24 ? 128 : 512;
  const uintll_t count_shards = count_messages / size_shards;
  const uintll_t bitcount_A = ceil(log((double)A)/log(2.0));;
  const uint_t count_counts = n + bitcount_A + 1;
//  const uint_t A=63877;//233;//641;

  uintll_t** dcounts;
  uintll_t** hcounts;
  dcounts = new uintll_t*[nr_dev];
  hcounts = new uintll_t*[nr_dev];
#pragma omp parallel for num_threads(nr_dev) schedule(static,1)
  for(int dev=0; dev<nr_dev; ++dev)
  {
    dim3 threads(128, 1, 1);
    uint_t xblocks;
    uintll_t offset, end;
    dim3 blocks;

    CHECK_ERROR( hipSetDevice(dev) );
    CHECK_ERROR( hipGetDeviceProperties(&prop, dev) );
    CHECK_ERROR( hipMalloc(dcounts+dev, count_counts*sizeof(uintll_t)) );
    CHECK_ERROR( hipMemset(dcounts[dev], 0, count_counts*sizeof(uintll_t)) );

    hcounts[dev] = new uintll_t[count_counts];
    memset(hcounts[dev], 0, count_counts*sizeof(uintll_t));

    if(verbose)
      printf("%d/%d threads on %s.\n", omp_get_thread_num()+1, omp_get_num_threads(), prop.name);
    if(dev==0)
      results_gpu.start(i_runtime);

    end = 0;
    {
      offset = count_shards / nr_dev / nr_dev * (dev)*(dev);
      end = count_shards / nr_dev / nr_dev * (dev+1)*(dev+1);

      xblocks = ceil(sqrt(1.0*(end-offset) / threads.x)) ;
      blocks.x= xblocks; blocks.y = xblocks;
      // 3) Remainder of the slice
      if(verbose)
        printf("Dev %d: Blocks: %d %d, offset %llu, end %llu, end %llu\n", dev, blocks.x, blocks.y, offset, end, (threads.x-1+threads.x * ((xblocks-1) * (xblocks) + (xblocks-1)) + offset)*size_shards);
      //dim3 blocks( (count_shards / threads.x)/2, 2 );
      if(n<=8)
        dancoding<1,32><<< blocks, threads >>>(n,A,dcounts[dev], offset, end);
      else if(n<=16)
        dancoding<16,64><<< blocks, threads >>>(n,A,dcounts[dev], offset, end);
      else if(n<=24)
        dancoding<128,64><<< blocks, threads >>>(n,A,dcounts[dev], offset, end);
      else
        dancoding<512,64><<< blocks, threads >>>(n,A,dcounts[dev], offset, end);
          
      CHECK_LAST("Kernel failed.");
    }
    if(dev==0) results_gpu.stop(i_runtime);
  }

  CHECK_ERROR(
      hipMemcpy(hcounts[0], dcounts[0], count_counts*sizeof(uintll_t), hipMemcpyDefault)
      );
  // other devices sum up to [0]
  for(int dev=1; dev<nr_dev; ++dev)
  {
    CHECK_ERROR(
      hipMemcpy(hcounts[dev], dcounts[dev], count_counts*sizeof(uintll_t), hipMemcpyDefault)
      );
    for(uint_t i=0; i<count_counts; ++i)
      hcounts[0][i] += hcounts[dev][i];
    CHECK_ERROR( hipFree(dcounts[dev]) );
    delete[] hcounts[dev];
  }

  results_cpu.stop(i_totaltime);

  // results
  uint128_t counts[64] = {0};
  counts[0] = 1ull<<n;
  //
  for(uint_t i=1; i<count_counts; ++i)
  {
    counts[i] = hcounts[0][i]<<1;
  }
  
  if(minb!=nullptr && mincb!=nullptr)
  {
    *minb=0xFFFF;;
    *mincb=0xFFFFFFFF;
    for(uint_t i=1; i<count_counts/2; ++i)
    {
      if(counts[i]!=0 && counts[i]<*mincb)
      {
        *minb=i;
        *mincb=counts[i];
      }
    }
  }

  if(verbose || file_output)
    process_result_ancoding(counts, stats, n, A, file_output?"ancoding_gpu":nullptr);

  CHECK_ERROR( hipFree(dcounts[0]) );
  delete[] hcounts[0];
  delete[] hcounts;
  delete[] dcounts;
  CHECK_ERROR( hipDeviceReset() );

}
