#include "hip/hip_runtime.h"

#include "globals.h"
#include "algorithms.h"
#include "an_coding.h"
#include <helper.h>

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#include <iostream>
#include <ostream>


template<uintll_t ShardSize,uint_t CountCounts>
__global__
void dancoding_grid_1d(uintll_t n, uintll_t A, uintll_t* counts, uintll_t offset, uintll_t end, uintll_t iterations, double stepsize)
{
  uintll_t shardXid = threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + offset;
  if(shardXid>=end)
    return;

  uintll_t counts_local[CountCounts] = { 0 };

  uintll_t w = A * shardXid * ShardSize;
  uintll_t wend = A * (shardXid+1) * ShardSize;
  uintll_t v;
  uintll_t it = 0;

  for(;w<wend;w+=A)
  {
    for(it=0; it<iterations; ++it)
    {
      v = it*stepsize;
      v *= A;
      ++counts_local[ __popcll( w^v ) ];
    }
  }
  for(int c=0; c<CountCounts; ++c)
    atomicAdd(counts+c, counts_local[c]);
}

template<uintll_t ShardSize,uint_t CountCounts>
__global__
void dancoding_grid_2d(uintll_t n, uintll_t A, uintll_t* counts, uintll_t offset, uintll_t end, uintll_t iterations, double stepsize)
{
  uintll_t shardXid = threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + offset;
  if(shardXid>=end)
    return;

  uintll_t counts_local[CountCounts] = { 0 };
  uintll_t s = stepsize;
  uintll_t w = A * shardXid * ShardSize * s;
  uintll_t w_end = A * (shardXid+1) * ShardSize * s;
  uintll_t v;
  uintll_t it = 0;

  for( ; w<w_end; w += A*s)
  {
    for(it=0; it<iterations; ++it)
    {
      v = it*stepsize;
      v *= A;
      ++counts_local[ __popcll( w^v ) ];
    }    
  }
  for(int c=0; c<CountCounts; ++c)
    atomicAdd(counts+c, counts_local[c]);
}

/**
 * Caller for kernel 
 */
template<uintll_t N>
struct Caller1D
{
  template<typename T>
  void operator()(dim3 blocks, dim3 threads, uintll_t n, uintll_t A, uintll_t* counts, uintll_t offset, uintll_t end, uintll_t iterations, T stepsize){
    dancoding_grid_1d<ANCoding::traits::Shards<N>::value, ANCoding::traits::CountCounts<N>::value ><<< blocks, threads >>>(n, A, counts, offset, end, iterations, stepsize);
  }
};
template<uintll_t N>
struct Caller2D
{
  template<typename T>
  void operator()(dim3 blocks, dim3 threads, uintll_t n, uintll_t A, uintll_t* counts, uintll_t offset, uintll_t end, uintll_t iterations, T stepsize){
    dancoding_grid_2d<ANCoding::traits::Shards<N>::value, ANCoding::traits::CountCounts<N>::value ><<< blocks, threads >>>(n, A, counts, offset, end, iterations, stepsize);
  }
};

double run_ancoding_grid(int gdim, uintll_t n, uintll_t iterations, uintll_t A, int verbose, double* times, uintll_t* minb, uintll_t* mincb, int file_output, int nr_dev_max)
{
  assert(A<(1ull<<n));
  int tmp_nr_dev;
  Statistics stats;
  TimeStatistics results_cpu (&stats,CPU_WALL_TIME);
  TimeStatistics results_gpu (&stats,GPU_TIME);
  int i_runtime = results_gpu.add("Kernel Runtime", "s");
  int i_totaltime = results_cpu.add("Total Runtime", "s");
  results_cpu.setFactorAll(0.001);
  results_gpu.setFactorAll(0.001);

  CHECK_ERROR( hipGetDeviceCount(&tmp_nr_dev) );
  const int nr_dev = nr_dev_max==0 ? tmp_nr_dev : min(nr_dev_max,tmp_nr_dev);
  hipDeviceProp_t prop;
  CHECK_ERROR( hipGetDeviceProperties(&prop, 0));
  if(verbose){
    printf("Start AN-Coding Algorithm - %dD Grid with %zu points\n", gdim, iterations);
    printf("Found %d CUDA devices (%s).\n", nr_dev, prop.name);
  }
  // skip init time
  for(int dev=0; dev<nr_dev; ++dev)
  {
    CHECK_ERROR( hipSetDevice(dev) );
    CHECK_ERROR( hipDeviceSynchronize() );
  }

  results_cpu.start(i_totaltime);

  uint_t n_up = n<=8 ? 8 : n<=16 ? 16 : n<=32 ? 32 : n<=40 ? 40 : 48;
  const uintll_t count_messages = (1ull << n);
  const uintll_t size_shards = ANCoding::getShardsSize(n_up); 
  iterations = iterations>count_messages?count_messages:iterations;

  const uintll_t count_shards = (gdim==2 ? iterations : count_messages) / size_shards;
  const uintll_t bitcount_A = ceil(log((double)A)/log(2.0));
  const uint_t count_counts = n + bitcount_A + 1;

  uintll_t** dcounts;
  uintll_t** hcounts;

  dcounts = new uintll_t*[nr_dev];
  hcounts = new uintll_t*[nr_dev];

#pragma omp parallel for num_threads(nr_dev) schedule(static,1)
  for(int dev=0; dev<nr_dev; ++dev)
  {
    dim3 threads(128, 1, 1);
    uint_t xblocks;
    uintll_t offset, end;
    dim3 blocks;

    CHECK_ERROR( hipSetDevice(dev) );
    CHECK_ERROR( hipGetDeviceProperties(&prop, dev) );
    CHECK_ERROR( hipMalloc(dcounts+dev, count_counts*sizeof(uintll_t)) );
    CHECK_ERROR( hipMemset(dcounts[dev], 0, count_counts*sizeof(uintll_t)) );

    hcounts[dev] = new uintll_t[count_counts];
    memset(hcounts[dev], 0, count_counts*sizeof(uintll_t));

    //end = 0;
    //    offset = count_shards / nr_dev / nr_dev * (dev)*(dev);
    //    end = count_shards / nr_dev / nr_dev * (dev+1)*(dev+1);
    offset = count_shards / nr_dev * dev;
    end = count_shards / nr_dev * (dev+1);

    xblocks = ceil(sqrt(1.0*(end-offset) / threads.x)) ;
    blocks.x= xblocks; blocks.y = xblocks;

    // 3) Remainder of the slice
    if(verbose){
      printf("%d/%d threads on %s.\n", omp_get_thread_num()+1, omp_get_num_threads(), prop.name);
      printf("Dev %d: Blocks: %d %d, offset %llu, end %llu, end %llu\n", dev, blocks.x, blocks.y, offset, end, (threads.x-1+threads.x * ((xblocks-1) * (xblocks) + (xblocks-1)) + offset)*size_shards);
    }
    if(dev==0)
      results_gpu.start(i_runtime);

    if(gdim==1)
      ANCoding::bridge<Caller1D>(n_up, blocks, threads, n, A, dcounts[dev], offset, end, iterations, 1.0L*count_messages/iterations);
    else
      ANCoding::bridge<Caller2D>(n_up, blocks, threads, n, A, dcounts[dev], offset, end, iterations, 1.0L*count_messages/iterations);

    CHECK_LAST("Kernel failed.");

    if(dev==0) results_gpu.stop(i_runtime);

  }

  CHECK_ERROR(
      hipMemcpy(hcounts[0], dcounts[0], count_counts*sizeof(uintll_t), hipMemcpyDefault)
      );
  // other devices sum up to [0]
  for(int dev=1; dev<nr_dev; ++dev)
  {
    CHECK_ERROR(
      hipMemcpy(hcounts[dev], dcounts[dev], count_counts*sizeof(uintll_t), hipMemcpyDefault)
      );
    for(uint_t i=0; i<count_counts; ++i)
      hcounts[0][i] += hcounts[dev][i];
    CHECK_ERROR( hipFree(dcounts[dev]) );
    delete[] hcounts[dev];
  }

  results_cpu.stop(i_totaltime);

  // results
  uint128_t counts[64] = {0};
//  counts[0] = 1ull<<n;
  for(uint_t i=0; i<count_counts; ++i)
  {
    counts[i] = static_cast<uint128_t>(static_cast<long double>(pow(2.0,n)/iterations*hcounts[0][i]));
    //<<1;//only <<1 if sorted
  }

  CHECK_ERROR( hipFree(dcounts[0]) );
  delete[] hcounts[0];
  delete[] hcounts;
  delete[] dcounts;
  CHECK_ERROR( hipDeviceReset() );

  // compute max. relative error
  double max_abs_error = get_rel_error_AN(A, n, counts, 0);

  if(minb!=nullptr && mincb!=nullptr)
  {
    *minb=0xFFFF;;
    *mincb=static_cast<uintll_t>(-1);
    for(uint_t i=1; i<count_counts/2; ++i)
    {
      if(counts[i]!=0 && counts[i]<static_cast<uint128_t>(*mincb))
      {
        *minb=i;
        *mincb=counts[i];
      }
    }
  }

  if(verbose)
  {
    process_result_ancoding_mc(counts,stats,n,A,iterations,file_output?"ancoding_grid1d":nullptr);
  }

  if(times!=NULL)
  {
    times[0] = stats.getAverage(0);
    times[1] = stats.getAverage(1);
  }


  return max_abs_error;
}
