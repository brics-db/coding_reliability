#include "hip/hip_runtime.h"

#include "globals.h"
#include "an_coding.h"
#include "algorithms.h"
#include "rand_gen.cuh"

#include <helper.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#include <iostream>
#include <ostream>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ int popc(uintll_t v){ return __popcll(v); }
__device__ int popc(uint_t v){ return __popc(v); }

template<uintll_t ShardSize,uint_t CountCounts, typename T, typename RandGenType>
__global__
void dancoding_mc(T n, T A, uintll_t* counts, T offset, T end, RandGenType* state, T iterations, double p2n)
{
  uint_t tid = threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x);
  T shardXid = threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + offset;
  if(shardXid>=end)
    return;

  T counts_local[CountCounts] = { 0 };

  T w = A * shardXid * ShardSize;
  T wend = A * (shardXid+1) * ShardSize;
  T v;
  T it;
  RandGenType local_state = state[tid];
  for(;w<wend;w+=A)
  {
    for(it=0; it<iterations; ++it)
    {
      v = static_cast<T>( p2n * hiprand_uniform_double(&local_state));
      v *= A;
      ++counts_local[ popc( w^v ) ];
    }
  }
  for(int c=0; c<CountCounts; ++c)
    atomicAdd(counts+c, counts_local[c]);
  state[tid] = local_state;
}
/**
 * Caller for kernel 
 */
template<uintll_t N>
struct Caller
{
  template<typename RandGenType>
  void operator()(uintll_t n, dim3 blocks, dim3 threads, uintll_t A, uintll_t* counts, uintll_t offset, uintll_t end, RandGenType* states, uintll_t iterations){
    double p2n = pow(2.0,n);
    if((A<<n)<(1ull<<32))
      dancoding_mc<ANCoding::traits::Shards<N>::value, ANCoding::traits::CountCounts<N>::value >
        <<< blocks, threads >>>((uint_t)n, (uint_t)A, counts, (uint_t)offset, (uint_t)end, states, (uint_t)iterations, p2n);
   else
      dancoding_mc<ANCoding::traits::Shards<N>::value, ANCoding::traits::CountCounts<N>::value >
        <<< blocks, threads >>>(n, A, counts, offset, end, states, iterations, p2n);
  }
};

double run_ancoding_mc(uintll_t n, uintll_t iterations, uintll_t A, int verbose, double* times, uintll_t* minb, uintll_t* mincb, int file_output, int nr_dev_max)
{
  int tmp_nr_dev;
  Statistics stats;
  TimeStatistics results_cpu (&stats,CPU_WALL_TIME);
  TimeStatistics results_gpu (&stats,GPU_TIME);
  int i_runtime = results_gpu.add("Kernel Runtime", "s");
  int i_totaltime = results_cpu.add("Total Runtime", "s");
  results_cpu.setFactorAll(0.001);
  results_gpu.setFactorAll(0.001);


  CHECK_ERROR( hipGetDeviceCount(&tmp_nr_dev) );
  const int nr_dev = nr_dev_max==0 ? tmp_nr_dev : min(nr_dev_max,tmp_nr_dev);
  hipDeviceProp_t prop;
  CHECK_ERROR( hipGetDeviceProperties(&prop, 0));
  if(verbose>1){
    printf("Start AN-Coding Algorithm - Monte Carlo with %zu iterations\n", iterations);
    printf("Found %d CUDA devices (%s).\n", nr_dev, prop.name);
  }
  // skip init time
  for(int dev=0; dev<nr_dev; ++dev)
  {
    CHECK_ERROR( hipSetDevice(dev) );
    CHECK_ERROR( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );
    CHECK_ERROR( hipDeviceSynchronize() );
  }

  results_cpu.start(i_totaltime);

  const uintll_t count_messages = (1ull << n);
  const uintll_t size_shards = ANCoding::getShardsSize(n);
  const uintll_t count_shards = count_messages / size_shards;
  const uintll_t bitcount_A = ceil(log((double)A)/log(2.0));
  const uint_t count_counts = n + bitcount_A + 1;

  uintll_t** dcounts;
  uintll_t** hcounts;

  dcounts = new uintll_t*[nr_dev];
  hcounts = new uintll_t*[nr_dev];
  iterations = iterations>count_messages?count_messages:iterations;

#pragma omp parallel for num_threads(nr_dev) schedule(static,1)
  for(int dev=0; dev<nr_dev; ++dev)
  {
    dim3 threads(128, 1, 1);
    uint_t xblocks;
    uintll_t offset, end;
    dim3 blocks;
    RandGen<RAND_GEN> gen;

    CHECK_ERROR( hipSetDevice(dev) );
    CHECK_ERROR( hipGetDeviceProperties(&prop, dev) );
    CHECK_ERROR( hipMalloc(dcounts+dev, count_counts*sizeof(uintll_t)) );
    CHECK_ERROR( hipMemset(dcounts[dev], 0, count_counts*sizeof(uintll_t)) );

    hcounts[dev] = new uintll_t[count_counts];
    memset(hcounts[dev], 0, count_counts*sizeof(uintll_t));


    //end = 0;
    //    offset = count_shards / nr_dev / nr_dev * (dev)*(dev);
    //    end = count_shards / nr_dev / nr_dev * (dev+1)*(dev+1);
    offset = count_shards / nr_dev * dev;
    end = count_shards / nr_dev * (dev+1);

    xblocks = ceil(sqrt(1.0*(end-offset) / threads.x)) ;
    blocks.x= xblocks; blocks.y = xblocks;

    // 3) Remainder of the slice
    if(verbose>1){
      printf("%d/%d threads on %s.\n", omp_get_thread_num()+1, omp_get_num_threads(), prop.name);
      printf("Dev %d: Blocks: %d %d, offset %llu, end %llu, end %llu\n", dev, blocks.x, blocks.y, offset, end, (threads.x-1+threads.x * ((xblocks-1) * (xblocks) + (xblocks-1)) + offset)*size_shards);
    }

    /* random generator stuff */
    gen.init(blocks, threads, 1337+8137*xblocks*xblocks*threads.x*dev, 1, dev);
    //dim3 blocks( (count_shards / threads.x)/2, 2 );
    if(dev==0)
      results_gpu.start(i_runtime);

    ANCoding::bridge<Caller>(n, blocks, threads, A, dcounts[dev], offset, end, gen.devStates, iterations);

    CHECK_LAST("Kernel failed.");

    if(dev==0) results_gpu.stop(i_runtime);

    gen.free();
  }

  CHECK_ERROR(
      hipMemcpy(hcounts[0], dcounts[0], count_counts*sizeof(uintll_t), hipMemcpyDefault)
      );
  // other devices sum up to [0]
  for(int dev=1; dev<nr_dev; ++dev)
  {
    CHECK_ERROR(
      hipMemcpy(hcounts[dev], dcounts[dev], count_counts*sizeof(uintll_t), hipMemcpyDefault)
      );
    for(uint_t i=0; i<count_counts; ++i)
      hcounts[0][i] += hcounts[dev][i];
    CHECK_ERROR( hipFree(dcounts[dev]) );
    delete[] hcounts[dev];
  }

  results_cpu.stop(i_totaltime);

  // results
  uint128_t counts[64] = {0};
//  counts[0] = 1ull<<n;
  for(uint_t i=0; i<count_counts; ++i)
  {
    counts[i] = static_cast<uint128_t>(static_cast<long double>(pow(2.0,n)/iterations*hcounts[0][i]));
    //<<1;//only <<1 if sorted
  }

  CHECK_ERROR( hipFree(dcounts[0]) );
  delete[] hcounts[0];
  delete[] hcounts;
  delete[] dcounts;
  CHECK_ERROR( hipDeviceReset() );

  // compute max. relative error
  double max_abs_error = get_rel_error_AN(A, n, counts, 0);

  if(minb!=nullptr && mincb!=nullptr)
  {
    *minb=0xFFFF;;
    *mincb=static_cast<uintll_t>(-1);
    for(uint_t i=1; i<count_counts/2; ++i)
    {
      if(counts[i]!=0 && counts[i]<static_cast<uint128_t>(*mincb))
      {
        *minb=i;
        *mincb=counts[i];
      }
    }
  }

  if(verbose || file_output)
  {
    if(nr_dev==4)
      process_result_ancoding_mc(counts,stats,n,A,iterations,file_output?"ancoding_mc_4gpu":nullptr);
    else
      process_result_ancoding_mc(counts,stats,n,A,iterations,file_output?"ancoding_mc_gpu":nullptr);
  }

  if(times!=NULL)
  {
    times[0] = stats.getAverage(0);
    times[1] = stats.getAverage(1);
  }


  return max_abs_error;
}
