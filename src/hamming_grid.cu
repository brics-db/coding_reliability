#include "hip/hip_runtime.h"

#include "globals.h"
#include "algorithms.h"
#include "hamming.h"
#include <helper.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#include <iostream>
#include <ostream>
#include <type_traits>

template<uintll_t N>
__device__ inline uintll_t computeHamming(const uintll_t &value) {
  uintll_t hamming = 0;
  hamming |= (__popcll(value & 0x56AAAD5B) & 0x1);
  hamming |= (__popcll(value & 0x9B33366D) & 0x1) << 1;
  hamming |= (__popcll(value & 0xE3C3C78E) & 0x1) << 2;
  hamming |= (__popcll(value & 0x03FC07F0) & 0x1) << 3;
 if(N<16)
  return (value << 4) | hamming;
 else // >=16
  hamming |= (__popcll(value & 0x03FFF800) & 0x1) << 4;
 if(N<32)
  return (value << 5) | hamming;
 else // >=32
  hamming |= (__popcll(value & 0xFC000000) & 0x1) << 5;
  return (value << 6) | hamming;
}

template<uintll_t N, uintll_t ShardSize,int CountCounts>
__global__
void dhamming_grid_1d(uintll_t* counts, uintll_t offset, uintll_t end, double stepsize)
{
  uintll_t shardXid = threadIdx.x + blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + offset;
  if(shardXid>=end)
    return;

  uintll_t counts_local[CountCounts] = { 0 };
  uintll_t w = shardXid * ShardSize * (uintll_t)stepsize;
  uintll_t x;
  
  for(uintll_t k=0; k<ShardSize; ++k)
  {
    x = w;
    x += k*stepsize;
    ++counts_local[ __popcll( computeHamming<N>( x ) ) ];
  }

  for(int c=0; c<CountCounts; ++c)
    atomicAdd(counts+c, counts_local[c]);
}

/**
 * Caller for kernel 
 */
template<uintll_t N>
struct Caller
{
  template<typename T>
  void operator()(dim3 blocks, dim3 threads, uintll_t* counts, uintll_t offset, uintll_t end, T stepsize){
    dhamming_grid_1d<N, Hamming::traits::Shards<N>::value, Hamming::traits::CountCounts<N>::value ><<< blocks, threads >>>(counts, offset, end, stepsize);
  }
};


double run_hamming_grid(uintll_t n, int with_1bit, uintll_t iterations, int file_output, int nr_dev_max)
{
  int tmp_nr_dev;
  Statistics stats;
  TimeStatistics results_cpu (&stats,CPU_WALL_TIME);
  TimeStatistics results_gpu (&stats,GPU_TIME);
  int i_runtime = results_gpu.add("Kernel Runtime", "s");
  int i_totaltime = results_cpu.add("Total Runtime", "s");
  results_cpu.setFactorAll(0.001);
  results_gpu.setFactorAll(0.001);
  const int verbose = 1;

  CHECK_ERROR( hipGetDeviceCount(&tmp_nr_dev) );
  const int nr_dev = nr_dev_max==0 ? tmp_nr_dev : min(nr_dev_max,tmp_nr_dev);
  hipDeviceProp_t prop;
  CHECK_ERROR( hipGetDeviceProperties(&prop, 0));
  if(verbose){
    printf("Start 1D Hamming-Coding Algorithm - 1D Grid with %zu iterations\n", iterations);
    printf("Found %d CUDA devices (%s).\n", nr_dev, prop.name);
  }
  // skip init time
  for(int dev=0; dev<nr_dev; ++dev)
  {
    CHECK_ERROR( hipSetDevice(dev) );
    CHECK_ERROR( hipDeviceSynchronize() );
  }

  results_cpu.start(i_totaltime);

  const uintll_t count_messages = (1ull << n);
  const uintll_t size_shards = Hamming::getShardsSize(n); // also used in template kernel launch
  iterations = iterations>count_messages?count_messages:iterations;

  const uintll_t count_shards = iterations / size_shards; // parallelize over iterations number
  const uint_t h = ( n==8 ? 5 : (n<32?6:7) );
  const uintll_t bitcount_message = n + h;
  const uint_t count_counts = bitcount_message + 1;


  uintll_t** dcounts;
  uintll_t** hcounts;

  dcounts = new uintll_t*[nr_dev];
  hcounts = new uintll_t*[nr_dev];

#pragma omp parallel for num_threads(nr_dev) schedule(static,1)
  for(int dev=0; dev<nr_dev; ++dev)
  {
    dim3 threads(128, 1, 1);
    uint_t xblocks;
    uintll_t offset, end;
    dim3 blocks;

    CHECK_ERROR( hipSetDevice(dev) );
    CHECK_ERROR( hipGetDeviceProperties(&prop, dev) );
    CHECK_ERROR( hipMalloc(dcounts+dev, count_counts*sizeof(uintll_t)) );
    CHECK_ERROR( hipMemset(dcounts[dev], 0, count_counts*sizeof(uintll_t)) );

    hcounts[dev] = new uintll_t[count_counts];
    memset(hcounts[dev], 0, count_counts*sizeof(uintll_t));


    //end = 0;
    //    offset = count_shards / nr_dev / nr_dev * (dev)*(dev);
    //    end = count_shards / nr_dev / nr_dev * (dev+1)*(dev+1);
    offset = count_shards / nr_dev * dev;
    end = count_shards / nr_dev * (dev+1);

    xblocks = ceil(sqrt(1.0*(end-offset) / threads.x)) ;
    blocks.x= xblocks; blocks.y = xblocks;

    // 3) Remainder of the slice
    if(verbose){
      printf("%d/%d threads on %s.\n", omp_get_thread_num()+1, omp_get_num_threads(), prop.name);
      printf("Dev %d: Blocks: %d %d, offset %llu, end %llu, end %llu\n", dev, blocks.x, blocks.y, offset, end, (threads.x-1+threads.x * ((xblocks-1) * (xblocks) + (xblocks-1)) + offset)*size_shards);
    }

    if(dev==0)
      results_gpu.start(i_runtime);

    Hamming::bridge<Caller>(n, blocks, threads, dcounts[dev], offset, end, 1.0L*count_messages/iterations);

    CHECK_LAST("Kernel failed.");

    if(dev==0) results_gpu.stop(i_runtime);

  }

  CHECK_ERROR(
      hipMemcpy(hcounts[0], dcounts[0], count_counts*sizeof(uintll_t), hipMemcpyDefault)
      );
  // other devices sum up to [0]
  for(int dev=1; dev<nr_dev; ++dev)
  {
    CHECK_ERROR(
      hipMemcpy(hcounts[dev], dcounts[dev], count_counts*sizeof(uintll_t), hipMemcpyDefault)
      );
    for(uint_t i=0; i<count_counts; ++i)
      hcounts[0][i] += hcounts[dev][i];
    CHECK_ERROR( hipFree(dcounts[dev]) );
    delete[] hcounts[dev];
  }

  results_cpu.stop(i_totaltime);


  // results
  uint128_t counts[64] = {0};
  counts[0] = static_cast<uint128_t>(static_cast<long double>(pow(2.0,n)*(hcounts[0][0])));
  for(uint_t i=2; i<count_counts; i+=2)
  {
    counts[i] = static_cast<uint128_t>(static_cast<long double>(pow(2.0,n)*(hcounts[0][i]+hcounts[0][i-1])));
  }
  if(with_1bit)
  {  
    // 1-bit sphere  
    for (uint_t i = 1; i < count_counts; i+=2)
    {
      if(i+1<count_counts){
        counts[i] = uint128_t(i+1)*counts[i+1] + uint128_t(bitcount_message-i+1)*counts[i-1];
      }else
        counts[i] = uint128_t(bitcount_message-i+1)*counts[i-1];
    }
  }

  CHECK_ERROR( hipFree(dcounts[0]) );
  delete[] hcounts[0];
  delete[] hcounts;
  delete[] dcounts;
  CHECK_ERROR( hipDeviceReset() );
  
  double max_abs_error = get_abs_error_hamming(n, counts, 0, with_1bit, nullptr);

  if(verbose)
  {    
    process_result_hamming_mc(counts,stats,n,h,with_1bit,iterations,file_output?"hamming_mc":nullptr);
  }

  return max_abs_error;
}
